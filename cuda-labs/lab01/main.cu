#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <locale.h>
#include <iostream>
#include <iomanip>
#include <cmath>

using namespace std;


void device_info()
{
	hipDeviceProp_t prop;
	setlocale(LC_ALL, "RUS");

	int count;
	hipGetDeviceCount(&count);

	for (int i = 0; i < count; i++)
	{
		hipGetDeviceProperties(&prop, i);

		cout << "--- ����� ���������� �� ���������� " << i << '\n';
		cout << "���: " << prop.name << '\n';
		cout << "�������������� �����������: " << prop.major << '.' << prop.minor << '\n';
		cout << "�������� �������: " << prop.clockRate << '\n';
		cout << "���������� �����������: ";
		if (prop.deviceOverlap)
			cout << "���������\n";
		else
			cout << "���������\n";

		cout << "����-��� ���������� ����: ";
		if (prop.kernelExecTimeoutEnabled)
			cout << "�������\n";
		else
			cout << "��������\n";

		cout << "--- ���������� � ������ ��� ���������� " << i << '\n';
		cout << "����� ���������� ������: " << prop.totalGlobalMem << '\n';
		cout << "����� ����������� ������: " << prop.totalConstMem << '\n';
		cout << "������������ ���: " << prop.memPitch << '\n';
		cout << "������������ �������: " << prop.textureAlignment << '\n';

		cout << "--- ���� � ����������������� ��� ���-�� " << i << '\n';
		cout << "���-�� �����������������: " << prop.multiProcessorCount << '\n';
		cout << "����������� ������ �� ���� ��: " << prop.sharedMemPerBlock << '\n';
		cout << "���������� ��������� �� ���� ��: " << prop.regsPerBlock << '\n';
		cout << "���������� ����� � �����: " << prop.warpSize << '\n';
		cout << "����.���-�� ����� � �����: " << prop.maxThreadsPerBlock << '\n';
		cout << "����.���-�� ����� �� ����������: (" << prop.maxThreadsDim[0] << ", " << prop.maxThreadsDim[1] << ", " << prop.maxThreadsDim[2] << ")\n";
		cout << "������������ ������� �����: (" << prop.maxGridSize[0] << ", " << prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << ")\n";
	}
}

namespace AddNumbers
{
	__global__ void add(int a, int b, int *c)
	{
		*c = a + b;
	}

	void run()
	{
		int c;
		int *dev_c;
		hipMalloc((void**)&dev_c, sizeof(int));
		add<<< 1, 1 >>>(2, 7, dev_c);
		hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
		cout << "2 + 7 = " << c << '\n';
		hipFree(dev_c);
	}
}

namespace AddVectors
{
	const int N = 10;

	__global__ void add_block(int *a, int *b, int *c)
	{
		int tid = blockIdx.x;
		c[tid] = a[tid] + b[tid];
		printf("Block number %d write %d in array on %d position\n", tid, c[tid], tid);
	}

	void run_block()
	{
		int a[N], b[N], c[N];
		int *dev_a, *dev_b, *dev_c;

		hipMalloc((void**)&dev_a, N * sizeof(int));
		hipMalloc((void**)&dev_b, N * sizeof(int));
		hipMalloc((void**)&dev_c, N * sizeof(int));

		for (int i = 0; i < N; i++)
		{
			a[i] = i;
			b[i] = i * i;
		}

		hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

		add_block<<< N, 1 >>> (dev_a, dev_b, dev_c);

		hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

		for (int i = 0; i < N; i++)
			cout << a[i] << " + " << b[i] << " = " << c[i] << '\n';

		hipFree(dev_a);
		hipFree(dev_b);
		hipFree(dev_c);
	}


	__global__ void add_thread(int *a, int *b, int *c)
	{
		int tid = threadIdx.x;
		c[tid] = a[tid] + b[tid];
		printf("Thread number %d write %d in array on %d position\n", tid, c[tid], tid);
	}

	void run_thread()
	{
		int a[N], b[N], c[N];
		int *dev_a, *dev_b, *dev_c;

		hipMalloc((void**)&dev_a, N * sizeof(int));
		hipMalloc((void**)&dev_b, N * sizeof(int));
		hipMalloc((void**)&dev_c, N * sizeof(int));

		for (int i = 0; i < N; i++)
		{
			a[i] = i;
			b[i] = i * i;
		}

		hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
		add_thread<<< 1, N >>> (dev_a, dev_b, dev_c);

		hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

		for (int i = 0; i < N; i++)
			cout << a[i] << " + " << b[i] << " = " << c[i] << '\n';

		hipFree(dev_a);
		hipFree(dev_b);
		hipFree(dev_c);
	}
}


namespace CudaPi
{
	//��������� ����� pi, ������� �������������� �������� ���������� ������� 1
	using CUDA_FLOAT = float;
	const int GRID_SIZE = 256;
	const int BLOCK_SIZE = 256;

	__global__ void pi_kern(CUDA_FLOAT *res)
	{
		int n = threadIdx.x + blockIdx.x * BLOCK_SIZE;
		CUDA_FLOAT x0 = n * 1.f / (BLOCK_SIZE * GRID_SIZE);
		CUDA_FLOAT y0 = sqrtf(1 - x0 * x0);
		CUDA_FLOAT dx = 1.f / (1.f * BLOCK_SIZE * GRID_SIZE);
		CUDA_FLOAT s = 0;
		CUDA_FLOAT x1, y1;
		
		x1 = x0 + dx;
		y1 = sqrtf(1 - x1 * x1);
		s = (y0 + y1) * dx / 2.f;
		res[n] = s;
	}

	int run() 
	{
		CUDA_FLOAT pi = 0;
		CUDA_FLOAT *res_d;
		CUDA_FLOAT res[GRID_SIZE * BLOCK_SIZE];

		hipMalloc((void**)&res_d, sizeof(CUDA_FLOAT)*GRID_SIZE * BLOCK_SIZE);

		dim3 grid(GRID_SIZE);
		dim3 block(BLOCK_SIZE);

		pi_kern<<< grid, block >>>(res_d);

		hipDeviceSynchronize();

		hipMemcpy(&res, res_d, sizeof(CUDA_FLOAT) * GRID_SIZE * BLOCK_SIZE, hipMemcpyDeviceToHost); 
		hipFree(res_d);

		for (int i = 0; i < GRID_SIZE * BLOCK_SIZE; i++)
			pi += res[i];
		
		pi *= 4;
		cout << "PI = " << pi <<'\n';

		return 0;
	}
}

namespace PowerOfK
{
	const int K = 3;
	const int SIZE = 32;

	__global__ void pow(long long powers[SIZE], int base)
	{
		int id = threadIdx.x;
		for (int i = 0; i < id; i++)
			powers[id] *= base;
	}

	void run()
	{
		long long h_powers[SIZE];
		long long *d_powers;

		hipMalloc(reinterpret_cast<void **>(&d_powers), SIZE * sizeof(long long));
		
		for (int i = 0; i < SIZE; i++) {
			h_powers[i] = 1;
		}

		hipMemcpy(d_powers, h_powers, SIZE * sizeof(long long), hipMemcpyHostToDevice);

		pow<<<1, SIZE>>>(d_powers, K);

		hipMemcpy(h_powers, d_powers, SIZE * sizeof(long long), hipMemcpyDeviceToHost);
		
		for (int i = 0; i < SIZE; i++) {
			cout << "Power " << i << ": " << h_powers[i] << '\n';
		}

		hipFree(d_powers);
	}
}


namespace RowSum
{
	const int SIZE = 5;

	__global__ void row_sum(int *matrix, size_t pitch, int result[SIZE])
	{
		int *row = (int *)((char *)matrix + threadIdx.x * pitch);
		for (int i = 0; i < SIZE; i++) {
			result[threadIdx.x] += row[i];
		}
	}

	void run()
	{
		int h_matrix[SIZE][SIZE] =
		{
			{ 1, 2, 3, 4, 5 },
			{ 6, 7, 8, 9, 0 },
			{ 1, 2, 3, 4, 5 },
			{ 6, 7, 8, 9, 0 },
			{ 1, 2, 3, 4, 5 }
		},
			h_result[SIZE] = { 0 }; 
		int *d_result, *d_matrix;
		size_t pitch;

		hipMalloc(&d_result, SIZE * sizeof(int));
		hipMemcpy(d_result, h_result, SIZE * sizeof(int), hipMemcpyHostToDevice);

		hipMallocPitch(&d_matrix, &pitch, SIZE * sizeof(int), SIZE);
		hipMemcpy2D(d_matrix, pitch, h_matrix, SIZE * sizeof(int), SIZE * sizeof(int), SIZE, hipMemcpyHostToDevice);

		row_sum <<<1, SIZE>>> (d_matrix, pitch, d_result);

		hipMemcpy(h_result, d_result, SIZE * sizeof(int), hipMemcpyDeviceToHost);

		for (int i = 0; i < SIZE; i++) {
			cout << "Sum of " << i << " row: " << h_result[i] << '\n';
		}

		hipFree(d_result);
		hipFree(d_matrix);
	}
}


namespace MultiplyMatrix
{
	const int SIZE = 5;

	__global__ void row_sum(int *matrix, size_t pitch, int factor)
	{
		int *item = (int *)((char *)matrix + threadIdx.x * pitch) + threadIdx.y;
		*item *= factor;
	}

	void run()
	{
		int factor = 5;
		int h_matrix[SIZE][SIZE] =
		{
			{ 1, 2, 3, 4, 5 },
			{ 6, 7, 8, 9, 0 },
			{ 1, 2, 3, 4, 5 },
			{ 6, 7, 8, 9, 0 },
			{ 1, 2, 3, 4, 5 }
		};
		int *d_matrix;
		size_t pitch;

		hipMallocPitch(&d_matrix, &pitch, SIZE * sizeof(int), SIZE);
		hipMemcpy2D(d_matrix, pitch, h_matrix, SIZE * sizeof(int), SIZE * sizeof(int), SIZE, hipMemcpyHostToDevice);

		dim3 threads(SIZE, SIZE);
		row_sum <<<1, threads>>> (d_matrix, pitch, factor);

		hipMemcpy2D(h_matrix, SIZE * sizeof(int), d_matrix, pitch, SIZE * sizeof(int), SIZE, hipMemcpyDeviceToHost);

		for (int i = 0; i < SIZE; i++) {
			for (int j = 0; j < SIZE; j++)
			{
				cout << setw(5) << h_matrix[i][j];
			}
			cout << '\n';
		}

		hipFree(d_matrix);
	}
}


int main()
{
	//device_info();
	//AddNumbers::run();
	//AddVectors::run_block();
	//AddVectors::run_thread();
	//CudaPi::run();
	//PowerOfK::run();
	//RowSum::run();
	MultiplyMatrix::run();

	return 0;
}